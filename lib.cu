#include "hip/hip_runtime.h"
#include "lib.hpp"

#include "cpp_cuda_project/geometry.hpp"
#include "cpp_only_project/parse_input.hpp"

namespace serac {

  using axom::cpp_only_project::parse_vec3;
  using axom::cpp_only_project::parse_float;
  
  using axom::cpp_cuda_project::sphere;
  using axom::cpp_cuda_project::separation;
  using axom::cpp_cuda_project::intersecting;
  
  __global__ void kernel(sphere a, sphere b) {
    printf("inside serac's cuda kernel...\n");
    printf("%f\n", separation(a, b));
    printf("%d\n", intersecting(a, b));
  }

  void do_some_calculation_on_GPU() {
    sphere a{parse_vec3("0.0 0.0 0.0"), parse_float("1.0")};
    sphere b{{3.0f, 0.0f, 0.0f}, 1.0f};

    kernel<<<1,1>>>(a, b);
    hipDeviceSynchronize();
  }

}