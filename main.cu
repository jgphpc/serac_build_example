#include "hip/hip_runtime.h"
#include "core.hpp"

__global__ void some_kernel_that_uses_axom(double * ptr) {
  axom::transform(ptr[threadIdx.x]);
}

int main() {
  axom::foo_device();

  double * ptr;
  hipMalloc(&ptr, sizeof(double) * 64);
  some_kernel_that_uses_axom<<<1,64>>>(ptr);
  hipDeviceSynchronize();
}
