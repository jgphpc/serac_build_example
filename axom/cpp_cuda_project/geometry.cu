#include "hip/hip_runtime.h"
#include "cpp_cuda_project/geometry.hpp"

#include <cmath>

namespace axom::cpp_cuda_project {

  AXOM_HOST_DEVICE float separation(sphere a, sphere b) {
    auto delta = b.center - a.center;
    return axom::core_project::norm(delta) - (a.radius + b.radius); 
  };

}