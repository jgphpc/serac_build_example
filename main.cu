#include "hip/hip_runtime.h"
#if defined(SPH_HIP)
#include "hip/hip_runtime.h"
#endif
#include <cstdio>

#include "cpp_cuda_project/geometry.hpp"
#include "cpp_only_project/parse_input.hpp"

using axom::cpp_only_project::parse_vec3;
using axom::cpp_only_project::parse_float;

using axom::cpp_cuda_project::sphere;
using axom::cpp_cuda_project::separation;
using axom::cpp_cuda_project::intersecting;

__global__ void serac_kernel(sphere a, sphere b) {
  printf("inside serac's cuda kernel...\n");
  printf("%f\n", separation(a, b));
  printf("%d\n", intersecting(a, b));
}

int main() {
  sphere a{parse_vec3("0.0 0.0 0.0"), parse_float("1.0")};
  sphere b{{3.0f, 0.0f, 0.0f}, 1.0f};

#if defined(SPH_CUDA)
//#if defined(__HIPCC__)
  serac_kernel<<<1,1>>>(a, b);
  hipDeviceSynchronize();
#elif defined(SPH_HIP)
//#elif defined(__HIPCC__)
  hipLaunchKernelGGL(serac_kernel, 1, 1, 0, 0, a, b);
  hipDeviceSynchronize();
#endif
}
