#include "hip/hip_runtime.h"
#include "core.hpp"

#include <iostream>

__global__ void some_kernel(double * ptr) {
  axom::transform(ptr[threadIdx.x]);
}

namespace axom{
  
  void foo_device() {
  
    std::cout << "running calculation on device" << std::endl;
  
    double * ptr;
    hipMalloc(&ptr, sizeof(double) * 64);
    some_kernel<<<1,64>>>(ptr);
    hipDeviceSynchronize();
      
  }
}


