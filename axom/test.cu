#include "hip/hip_runtime.h"
#include <cstdio>

#include "cpp_cuda_project/geometry.hpp"

using axom::cpp_cuda_project::sphere;
using axom::cpp_cuda_project::separation;
using axom::cpp_cuda_project::intersecting;

__global__ void kernel(sphere a, sphere b) {
  printf("inside cuda kernel...\n");
  printf("%f\n", separation(a, b));
  printf("%d\n", intersecting(a, b));
}

int main() {
  sphere a{{0.0f, 0.0f, 0.0f}, 1.0f};
  sphere b{{3.0f, 0.0f, 0.0f}, 1.0f};

  kernel<<<1,1>>>(a, b);
  hipDeviceSynchronize();
}