#include "hip/hip_runtime.h"
#include "core.hpp"
#include "core.cuh"

#include <iostream>

__global__ void some_kernel(double * ptr) {
  transform(ptr[threadIdx.x]);
}

namespace axom{
  void foo_device() {
  
    std::cout << "running calculation on device" << std::endl;
  
    double * ptr;
    hipMalloc(&ptr, sizeof(double) * 64);
    some_kernel<<<1,64>>>(ptr);
    hipDeviceSynchronize();
      
  }
}


